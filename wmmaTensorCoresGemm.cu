#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// CUDA sample demonstrating a GEMM computation using the Warp Matrix Multiply
// and Accumulate API introduced in CUDA 9.

// In this program, the compute_gemm kernel computes the result of a matrix
// multiplication and addition: D = alpha * A * B + beta * C. The dimensions of
// both C and D matrices are M_GLOBAL x N_GLOBAL. The A matrix is M_GLOBAL x
// K_GLOBAL (row-major), the B matrix is K_GLOBAL x N_GLOBAL (column-major). In
// that kernel, each CTA computes one 128 x 128 tile of the resulting matrix per
// iteration. When the tile is computed, the CTA stores it to the global memory
// and begins a new iteration, selecting a new 128 x 128 tile to compute.
// Each CTA consists of eight warps. For the 128 x 128 tile, each warp computes
// eight 16 x 16 subtiles, organized in a 2 x 4 two-dimensional array. Warps
// compute the 16 x 16 subtiles using nvcuda::wmma::mma_sync operations by
// moving through the K_GLOBAL dimension of the A and B matrices and
// accumulating the intermediate result in the local thread state.

// There are a number of simple optimizations used in the algorithm:
// - The CTA copies the 128 x 128 tile of the C matrix from the global memory to
//   shared memory. After that is done, each warp loads the C matrix fragments
//   from shared memory, thus avoiding a random global memory access.
// - On each internal iteration, the CTA copies a portion of the A and B
//   matrices from global memory to shared memory. After that, all warps in the
//   CTA reuse the A and B data from shared memory, thus reducing the number of
//   data copies from global memory.
// - The portions of the A and B matrices are stored in shared memory with an
//   additional padding (skew) to reduce the number of shared memory access bank
//   conflicts.
//   (See a detailed explanation near the SKEW_HALF macro definition.)
// - When the CTA finishes computing the tiles of the resulting matrix, each
//   warp stores its subtiles to shared memory. The CTA then copies the shared
//   memory contents to global memory, again avoiding redundant random global
//   memory  accesses.
// - Note that the CTA tile size is chosen to maximize the GPU register
//   utilization, but carefully enough to avoid local memory use.

#include <assert.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <stdio.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include "params.h"
#include "wmmaTensorCoresGemm.cuh"

#if COMPUTE_PREC != 64

using namespace nvcuda;

__host__ void init_host_matrices(half *a, half *b, float *c) {
    for (unsigned  int i = 0; i < M_GLOBAL; i++) {
        for (unsigned int j = 0; j < K_GLOBAL; j++) {
            a[i * K_GLOBAL + j] = (half)(rand() % 3);
        }
    }

    for (unsigned int i = 0; i < N_GLOBAL; i++) {
        for (unsigned int j = 0; j < K_GLOBAL; j++) {
            b[i * K_GLOBAL + j] = (half)(rand() % 3);
        }
    }

    for (unsigned int t = 0; t < M_GLOBAL * N_GLOBAL; t++) {
        c[t] = static_cast<float>(rand() % 3);
    }
}

__global__ void compute_gemm(const half *A, const half *B, const float *C,
                             float *D, float alpha, float beta) {
    extern __shared__ half shmem[][CHUNK_K * K + SKEW_HALF];

    // Warp and lane identification.
    const unsigned int warpId = threadIdx.x / WARP_SIZE;
    const unsigned int laneId = threadIdx.x % WARP_SIZE;

    // Offset in shared memory from which the B matrix is stored.
    const size_t shmem_idx_b_off = BLOCK_COL_TILES * M;

    // This pointer is used to access the C and D matrix tiles this warp computes.
    float *shmem_warp_tile_ptr = (float *)&shmem[0][0] +
                                 (warpId / 2) * SHMEM_STRIDE * K * 2 +
                                 (warpId % 2) * SHMEM_OFFSET;

    // This pointer is used to stream the C and D matrices block-wide tile to and
    // from shared memory.
    float *shmem_warp_stream_ptr =
            (float *)&shmem[0][0] + warpId * SHMEM_STRIDE * K;

    // Adjust the beta scaler, as it'll be multiplied by alpha at the end of
    // each tile computation. Technically this is not generally correct (may
    // result in a loss of precision). Zero still needs to be specially handled
    // though.
    beta /= alpha;

    // Each CTA slides along the 128 x 128 tiles from the top left corner of the
    // matrix to the right and down, and selects the next tile to compute. Once
    // there's no such tile, all warps in this CTA exit.
    for (unsigned int block_pos = blockIdx.x;; block_pos += gridDim.x) {
        const unsigned int block_tile_i =
                ((block_pos * BLOCK_ROW_TILES) / N_TILES) * (BLOCK_COL_TILES);
        const unsigned int block_tile_j = (block_pos * BLOCK_COL_TILES) % N_TILES;

        // Stop when there are no more D matrix tiles to compute in this CTA.
        if (block_tile_i >= M_TILES) {
            break;
        }

        // This warp's pointer to the C matrix data to copy memory from to shared
        // memory.
        const size_t gmem_idx =
                (block_tile_i + warpId) * M * GLOBAL_MEM_STRIDE + block_tile_j * N;
        const float *src_gmem_warp_stream_ptr = &C[gmem_idx];

        // Stream multiple C tiles to shared memory.
#pragma unroll
        for (int i = 0; i < K; i++) {
            typedef int4 copy_t;

            *((copy_t *)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId) =
                    *((copy_t *)(src_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) +
                      laneId);
        }

        __syncthreads();

        // These fragments will accumulate the result of A and B matrix fragment
        // multiplications along the K_GLOBAL dimension.
        wmma::fragment<wmma::accumulator, M, N, K, float> c[WARP_COL_TILES]
        [WARP_ROW_TILES];

        // Load the C matrix tiles into fragments from shared memory.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
                const float *tile_ptr =
                        shmem_warp_tile_ptr + i * SHMEM_STRIDE * K + j * N;

                wmma::load_matrix_sync(c[i][j], tile_ptr, SHMEM_STRIDE, C_LAYOUT);
            }
        }

        __syncthreads();

        // Scale the C matrix.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
                for (int t = 0; t < c[i][j].num_elements; t++) {
                    c[i][j].x[t] *= beta;
                }
            }
        }

        // Select what warp copies what matrix to shared memory.
        // Warps 0-3 copy the A matrix, warps 4-7 copy the B matrix.
        const half *warp_ptr = (warpId < 4) ? (&A[block_tile_i * M * K_GLOBAL] +
                                               M * K_GLOBAL * (warpId % 4) * 2)
                                            : (&B[block_tile_j * N * K_GLOBAL] +
                                               N * K_GLOBAL * (warpId % 4) * 2);

        // Go through the global K dimension by a fixed step at a time.
#pragma unroll
        for (int tile_k = 0; tile_k < K_TILES; tile_k += CHUNK_K) {
            // Copy slices of the A and B matrices to shared memory.
            // The first half of the warps in the CTA copy the A matrix, the rest copy
            // the B matrix.
            size_t shmem_idx =
                    warpId < (WARPS_PER_BLOCK / 2)
                    ? (M * (warpId % (WARPS_PER_BLOCK / 2)) * 2)
                    : (N * (warpId % (WARPS_PER_BLOCK / 2)) * 2 + shmem_idx_b_off);

            // First half of the warp copies the first row / column of the matrix,
            // the second half of the warp copies the next.
            int4 *lane_ptr = (int4 *)(warp_ptr + tile_k * K +
                                      (laneId / CHUNK_COPY_LINE_LANES) * K_GLOBAL) +
                             (laneId % CHUNK_COPY_LINE_LANES);

            // Shift the second half of the warp to the next row / column in the
            // shared memory.
            shmem_idx += laneId / CHUNK_COPY_LINE_LANES;

#pragma unroll
            for (int i = 0; i < ((WARP_SIZE / 2) / CHUNK_COPY_LINES_PER_WARP) * 2;
                 i++) {
                // Copy 16 bytes at once in each lane.
                *((int4 *)&shmem[shmem_idx][0] + (laneId % CHUNK_COPY_LINE_LANES)) =
                        *lane_ptr;

                // Advance the global memory pointer and the shared memory index.
                lane_ptr =
                        (int4 *)((half *)lane_ptr + K_GLOBAL * CHUNK_COPY_LINES_PER_WARP);
                shmem_idx += CHUNK_COPY_LINES_PER_WARP;
            }

            __syncthreads();

            // Compute a grid of C matrix tiles in each warp.
#pragma unroll
            for (int k_step = 0; k_step < CHUNK_K; k_step++) {
                wmma::fragment<wmma::matrix_a, M, N, K, half, wmma::row_major>
                        a[WARP_COL_TILES];
                wmma::fragment<wmma::matrix_b, M, N, K, half, wmma::col_major>
                        b[WARP_ROW_TILES];

#pragma unroll
                for (int i = 0; i < WARP_COL_TILES; i++) {
                    size_t shmem_idx_a = (warpId / 2) * M * 2 + (i * M);
                    const half *tile_ptr = &shmem[shmem_idx_a][k_step * K];

                    wmma::load_matrix_sync(a[i], tile_ptr, K * CHUNK_K + SKEW_HALF);

#pragma unroll
                    for (int j = 0; j < WARP_ROW_TILES; j++) {
                        if (i == 0) {
                            // Load the B matrix fragment once, because it is going to be
                            // reused against the other A matrix fragments.
                            size_t shmem_idx_b = shmem_idx_b_off +
                                                 (WARP_ROW_TILES * N) * (warpId % 2) +
                                                 (j * N);
                            const half *tile_ptr = &shmem[shmem_idx_b][k_step * K];

                            wmma::load_matrix_sync(b[j], tile_ptr, K * CHUNK_K + SKEW_HALF);
                        }

                        wmma::mma_sync(c[i][j], a[i], b[j], c[i][j]);
                    }
                }
            }

            __syncthreads();
        }

        // Store the D fragments to shared memory.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
                // Uniform, point-wise transformations of ALL fragment elements by ALL
                // threads in the warp are well-defined even though element indices
                // within fragment storage are not defined.
                for (int t = 0; t < c[i][j].num_elements; t++) c[i][j].x[t] *= alpha;

                float *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * K + j * N;

                wmma::store_matrix_sync(tile_ptr, c[i][j], SHMEM_STRIDE, C_LAYOUT);
            }
        }

        __syncthreads();

        // Now that shared memory contains all the D tiles, stream them to global
        // memory.
        float *dst_gmem_warp_stream_ptr = &D[gmem_idx];

#pragma unroll
        for (int i = 0; i < K; i++) {
            *((int4 *)(dst_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) + laneId) =
                    *((int4 *)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId);
        }

        __syncthreads();
    }
}

// Performs an MxNxK GEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 16.
//  3) Neither A nor B are transposed.
// Note: This is a less performant version of the compute_gemm kernel. It is
// designed for
//       demonstration purposes only to show the CUDA WMMA API use without
//       relying on availability of the shared memory.
__global__ void simple_wmma_gemm(half *a, half *b, float *c, float *d, int m_ld,
                                 int n_ld, int k_ld, float alpha, float beta) {
    // Leading dimensions. Packed with no transpositions.
    int lda = k_ld;
    int ldb = k_ld;
    int ldc = n_ld;

    // Tile using a 2D grid
    int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
    int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

    // Declare the fragments
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major>
            a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major>
            b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

    wmma::fill_fragment(acc_frag, 0.0f);

    // Loop over k
    for (int i = 0; i < k_ld; i += WMMA_K) {
        int aCol = i;
        int aRow = warpM * WMMA_M;
        int bCol = warpN * N;
        int bRow = i;

        // Bounds checking
        if (aRow < m_ld && aCol < k_ld && bRow < k_ld && bCol < n_ld) {
            // Load the inputs
            wmma::load_matrix_sync(a_frag, a + aCol + aRow * lda, lda);
            wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);

            // Perform the matrix multiplication
            wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
        }
    }

    // Load in the current value of c, scale it by beta, and add this our result
    // scaled by alpha
    int cCol = warpN * WMMA_N;
    int cRow = warpM * WMMA_M;

    if (cRow < m_ld && cCol < n_ld) {
        wmma::load_matrix_sync(c_frag, c + cCol + cRow * ldc, ldc,
                               wmma::mem_row_major);

        for (int i = 0; i < c_frag.num_elements; i++) {
            c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
        }

        // Store the output
        wmma::store_matrix_sync(d + cCol + cRow * ldc, c_frag, ldc,
                                wmma::mem_row_major);
    }
}

__host__ void matMultiplyOnHost(half *A, half *B, float *C, float alpha,
                                float beta, int numARows, int numAColumns,
                                int numBRows, int numBColumns, int numCRows,
                                int numCColumns) {
    for (int i = 0; i < numCRows; i++) {
        for (int j = 0; j < numCColumns; j++) {
            float temp = 0.0;

            for (int k = 0; k < numAColumns; k++) {
                temp += (float)A[i * numAColumns + k] * (float)B[j * numBRows + k];
            }

            C[i * numCColumns + j] = temp * alpha + beta * C[i * numCColumns + j];
        }
    }
}

#endif // COMPUTE_PREC != 64