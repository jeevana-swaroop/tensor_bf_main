#include "hip/hip_runtime.h"
#include "dmmaTensorCoresGemm.cuh"
#include "params.h"

/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


// CUDA sample demonstrating a Double precision GEMM computation using the Warp
//  Matrix Multiply and Accumulate API introduced in CUDA 11.0.

// In this program, the compute_dgemm kernel computes the result of a matrix multiplication
// and addition: D = alpha * A * B + beta * C. The dimensions of both C and D matrices
// are M_GLOBAL x N_GLOBAL. The A matrix is M_GLOBAL x K_GLOBAL (row-major), the B matrix
// is K_GLOBAL x N_GLOBAL (column-major).
// In that kernel, each CTA computes one 64 x 64 tile of the resulting matrix
// per iteration. When the tile is computed, the CTA stores it to the global memory
// and begins a new iteration, selecting a new 64 x 64 tile to compute.
// Each CTA consists of eight warps. For the 64 x 64 tile, each warp computes eight
// 8 x 8 subtiles, organized in a 2 x 4 two-dimensional array.
// Warps compute the 8 x 8 subtiles using nvcuda::wmma::mma_sync operations by
// moving through the K_GLOBAL dimension of the A and B matrices and accumulating
// the intermediate result in the local thread state.

// There are a number of simple optimizations used in the algorithm:
// - The CTA copies the 64 x 64 tile of the C matrix from the global memory to
//   shared memory. After that is done, each warp loads the C matrix fragments from
//   shared memory, thus avoiding a random global memory access.
// - On each internal iteration, the CTA copies a portion of the A and B matrices from
//   global memory to shared memory. After that, all warps in the CTA reuse the A and B
//   data from shared memory, thus reducing the number of data copies from global memory.
// - The portions of the A and B matrices are stored in shared memory with an additional
//   padding (skew) to reduce the number of shared memory access bank conflicts.
//   (See a detailed explanation near the SKEW_DOUBLE macro definition.)
// - When the CTA finishes computing the tiles of the resulting matrix, each warp stores
//   its subtiles to shared memory. The CTA then copies the shared memory contents to
//   global memory, again avoiding redundant random global memory accesses.
// - Note that the CTA tile size is chosen to maximize the GPU register utilization,
//   but carefully enough to avoid local memory use.

#include <assert.h>
#include <stdio.h>
//#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_cooperative_groups.h>
//#include <cooperative_groups/memcpy_async.h>
#include <cuda/std/type_traits>
#include <cuda/barrier>
//#include <cuda/pipeline>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#if COMPUTE_PREC == 64

enum kernels
{
    dmma_shmem_gemm_async_copy      = 0, // DMMA shmem using kernel with async_copy
    dmma_shmem_gemm_cg_async_copy   = 1, // DMMA shmem using kernel with cooperative groups async_copy
    dmma_shmem_gemm                 = 2, // DMMA shmem using kernel normal copy (without async_copy).
    simple_dmma_gemm                = 3  // DMMA non-shmem using simple kernel.
};

const char* kernelNames[] = {"compute_dgemm_async_copy", "compute_dgemm_cg_async_copy",
                             "compute_dgemm", "simple_wmma_gemm"};

using namespace nvcuda;
namespace cg = cooperative_groups;

__host__ void init_host_matrices(double *a, double *b, double *c)
{
    for (unsigned int i = 0; i < M_GLOBAL; i++) {
        for (unsigned int j = 0; j < K_GLOBAL; j++) {
            a[i*K_GLOBAL+j] = (double) (rand() % 3);
        }
    }

    for (unsigned int i = 0; i < N_GLOBAL; i++) {
        for (unsigned int j = 0; j < K_GLOBAL; j++) {
            b[i*K_GLOBAL+j] = (double) (rand() % 3);
        }
    }

    for (unsigned int t = 0; t < M_GLOBAL * N_GLOBAL; t++) {
        c[t] =  (double) (rand() % 3);
    }
}

__global__ void compute_dgemm(const double *A, const double *B, const double *C, double *D, double alpha, double beta)
{
#if __CUDA_ARCH__ >= 800
    extern __shared__ double shmem[][CHUNK_K * K + SKEW_DOUBLE];

    // Warp and lane identification.
    const unsigned int warpId = threadIdx.x / WARP_SIZE;
    const unsigned int laneId = threadIdx.x % WARP_SIZE;

    // Offset in shared memory from which the B matrix is stored.
    const size_t shmem_idx_b_off = BLOCK_COL_TILES * M;


    // This pointer is used to access the C and D matrix tiles this warp computes.
    double *shmem_warp_tile_ptr = (double*)&shmem[0][0] + (warpId / BLOCK_ROW_WARPS) * SHMEM_STRIDE * N * BLOCK_ROW_WARPS + (warpId % BLOCK_ROW_WARPS) * SHMEM_OFFSET;

    // This pointer is used to stream the C and D matrices block-wide tile to and from shared memory.
    double *shmem_warp_stream_ptr = (double*)&shmem[0][0] + warpId * SHMEM_STRIDE * N;

    // Adjust the beta scaler, as it'll be multiplied by alpha at the end of
    // each tile computation. Technically this is not generally correct (may result
    // in a loss of precision). Zero still needs to be specially handled though.
    beta /= alpha;

    // Each CTA slides along the 64 x 64 tiles from the top left corner of the matrix to the
    // right and down, and selects the next tile to compute. Once there's no such tile,
    // all warps in this CTA exit.
    for(unsigned int block_pos = blockIdx.x;; block_pos += gridDim.x) {
        const unsigned int block_tile_i = ((block_pos * BLOCK_ROW_TILES) / N_TILES) * (BLOCK_COL_TILES);
        const unsigned int block_tile_j = (block_pos * BLOCK_COL_TILES) % N_TILES;

        // Stop when there are no more D matrix tiles to compute in this CTA.
        if (block_tile_i >= M_TILES) {
            break;
        }

        // This warp's pointer to the C matrix data to copy memory from to shared memory.
        const size_t gmem_idx = (block_tile_i + warpId) * M * GLOBAL_MEM_STRIDE + block_tile_j * N;
        const double *src_gmem_warp_stream_ptr = &C[gmem_idx];

        // Stream multiple C tiles to shared memory.
#pragma unroll
        for (int i = 0; i < N; i++) {
            *((int4 *)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId) =
                *((int4 *)(src_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) + laneId);
        }

        __syncthreads();

        // These fragments will accumulate the result of A and B matrix fragment multiplications
        // along the K_GLOBAL dimension.
        wmma::fragment<wmma::accumulator, M, N, K, double> c[WARP_COL_TILES][WARP_ROW_TILES];

        // Load the C matrix tiles into fragments from shared memory.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
                const double *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * N + j * N;

                wmma::load_matrix_sync(c[i][j], tile_ptr, SHMEM_STRIDE, C_LAYOUT);
            }
        }

        __syncthreads();

        // Scale the C matrix.
#pragma unroll
       for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
                for (int t = 0; t < c[i][j].num_elements; t++) {
                    c[i][j].x[t] *= beta;
                }
            }
        }

        // Select what warp copies what matrix to shared memory.
        // Warps 0-3 copy the A matrix, warps 4-7 copy the B matrix.
        const double *warp_ptr = (warpId < (WARPS_PER_BLOCK/2)) ? (&A[block_tile_i * M * K_GLOBAL] + M * K_GLOBAL * (warpId % (WARPS_PER_BLOCK/2)) * 2) :
                                              (&B[block_tile_j * N * K_GLOBAL] + N * K_GLOBAL * (warpId % (WARPS_PER_BLOCK/2)) * 2);

        // Go through the global K dimension by a fixed step at a time.
#pragma unroll
        for (int tile_k = 0; tile_k < K_TILES; tile_k += CHUNK_K) {
            // Copy slices of the A and B matrices to shared memory.
            // The first half of the warps in the CTA copy the A matrix, the rest copy the B matrix.
            size_t shmem_idx = warpId < (WARPS_PER_BLOCK/2) ? (M * (warpId % (WARPS_PER_BLOCK/2)) * 2) :
                                                              (N * (warpId % (WARPS_PER_BLOCK/2)) * 2 + shmem_idx_b_off);

            // First half of the warp copies the first row / column of the matrix,
            // the second half of the warp copies the next.
            const double *lane_ptr = warp_ptr + tile_k * K + (laneId / CHUNK_COPY_LINE_LANES) * K_GLOBAL;

            // Shift the second half of the warp to the next row / column in the shared memory.
            shmem_idx += laneId / CHUNK_COPY_LINE_LANES;

#pragma unroll
            for(int i = 0; i < ((WARP_SIZE/2) / CHUNK_COPY_LINES_PER_WARP); i++) {
                 // Copy 16 bytes at once in each lane.
                *((int4*)&shmem[shmem_idx][0] + (laneId % CHUNK_COPY_LINE_LANES)) = *((int4*)lane_ptr +  (laneId % CHUNK_COPY_LINE_LANES));

                // Advance the global memory pointer and the shared memory index.
                lane_ptr = lane_ptr + K_GLOBAL * CHUNK_COPY_LINES_PER_WARP;
                shmem_idx += CHUNK_COPY_LINES_PER_WARP;
            }

            __syncthreads();

            // Compute a grid of C matrix tiles in each warp.
#pragma unroll
            for (int k_step = 0; k_step < CHUNK_K; k_step++) {
                wmma::fragment<wmma::matrix_a, M, N, K, double, wmma::row_major> a[WARP_COL_TILES];
                wmma::fragment<wmma::matrix_b, M, N, K, double, wmma::col_major> b[WARP_ROW_TILES];

#pragma unroll
                for (int i = 0; i < WARP_COL_TILES; i++) {
                    size_t shmem_idx_a = (warpId/2) * M * 2 + (i * M);
                    const double *tile_ptr = &shmem[shmem_idx_a][k_step * K];

                    wmma::load_matrix_sync(a[i], tile_ptr, K * CHUNK_K + SKEW_DOUBLE);

#pragma unroll
                    for (int j = 0; j < WARP_ROW_TILES; j++) {
                        if (i == 0) {
                            // Load the B matrix fragment once, because it is going to be reused
                            // against the other A matrix fragments.
                            size_t shmem_idx_b = shmem_idx_b_off + (WARP_ROW_TILES * N) * (warpId%2) + (j * N);
                            const double *tile_ptr = &shmem[shmem_idx_b][k_step * K];

                            wmma::load_matrix_sync(b[j], tile_ptr, K * CHUNK_K + SKEW_DOUBLE);

                        }

                        wmma::mma_sync(c[i][j], a[i], b[j], c[i][j]);
                    }
                }
            }

            __syncthreads();
        }

        // Store the D fragments to shared memory.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
                // Uniform, point-wise transformations of ALL fragment elements by ALL threads in the
                // warp are well-defined even though element indices within fragment storage are not defined.
#pragma unroll
                for (int t = 0; t < c[i][j].num_elements; t++)
                    c[i][j].x[t] *= alpha;

                double *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * N + j * N;

                wmma::store_matrix_sync(tile_ptr, c[i][j], SHMEM_STRIDE, C_LAYOUT);
            }
        }

        __syncthreads();

        // Now that shared memory contains all the D tiles, stream them to global memory.
        double *dst_gmem_warp_stream_ptr = &D[gmem_idx];

#pragma unroll
        for (int i = 0; i < N; i++) {
            *((int4*)(dst_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) + laneId) =
                *((int4*)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId);
        }

        __syncthreads();
    }
#endif
}

//__global__ void compute_dgemm_async_copy(const double *A, const double *B, const double *C, double *D, double alpha, double beta)
//{
//#if __CUDA_ARCH__ >= 800
//    extern __shared__ double shmem[][CHUNK_K * K + SKEW_DOUBLE];
//
//    // Warp and lane identification.
//    const unsigned int warpId = threadIdx.x / WARP_SIZE;
//    const unsigned int laneId = threadIdx.x % WARP_SIZE;
//
//    // Offset in shared memory from which the B matrix is stored.
//    constexpr size_t shmem_idx_b_off = BLOCK_COL_TILES * M;
//
//    // This pointer is used to access the C and D matrix tiles this warp computes.
//    double *shmem_warp_tile_ptr = &shmem[0][0] + (warpId/BLOCK_ROW_WARPS) * SHMEM_STRIDE * N * BLOCK_ROW_WARPS + (warpId % BLOCK_ROW_WARPS) * SHMEM_OFFSET;
//
//    // This pointer is used to stream the C and D matrices block-wide tile to and from shared memory.
//    double *shmem_warp_stream_ptr = &shmem[0][0] + warpId * SHMEM_STRIDE * N;
//
//    // Adjust the beta scaler, as it'll be multiplied by alpha at the end of
//    // each tile computation. Technically this is not generally correct (may result
//    // in a loss of precision). Zero still needs to be specially handled though.
//    beta /= alpha;
//
//    cuda::pipeline<cuda::thread_scope_thread> pipe = cuda::make_pipeline();
//
//    const auto shape2 = cuda::aligned_size_t<alignof(double2)>(sizeof(double2));
//    constexpr int loadStride = 1; // load 2 double, left-shift by 1.
//
//    // Each CTA slides along the 64 x 64 tiles from the top left corner of the matrix to the
//    // right and down, and selects the next tile to compute. Once there's no such tile,
//    // all warps in this CTA exit.
//    for(unsigned int block_pos = blockIdx.x;; block_pos += gridDim.x) {
//        const unsigned int block_tile_i = ((block_pos * BLOCK_ROW_TILES) / N_TILES) * (BLOCK_COL_TILES);
//        const unsigned int block_tile_j = (block_pos * BLOCK_COL_TILES) % N_TILES;
//
//        // Stop when there are no more D matrix tiles to compute in this CTA.
//        if (block_tile_i >= M_TILES) {
//            break;
//        }
//
//        // This warp's pointer to the C matrix data to copy memory from to shared memory.
//        const size_t gmem_idx = (block_tile_i + warpId) * M * GLOBAL_MEM_STRIDE + block_tile_j * N;
//        const double *src_gmem_warp_stream_ptr = &C[gmem_idx];
//
//        // Stream multiple C tiles to shared memory.
//#pragma unroll
//        for (int i = 0; i < N; i++) {
//            pipe.producer_acquire();
//            cuda::memcpy_async(&shmem_warp_stream_ptr[(SHMEM_STRIDE * i) + (laneId << loadStride)],
//                                &src_gmem_warp_stream_ptr[(GLOBAL_MEM_STRIDE * i) + (laneId << loadStride)],
//                                shape2, pipe);
//
//            pipe.producer_commit();
//        }
//        // Now wait for all the above issued 8 batches to complete.
//        cuda::pipeline_consumer_wait_prior<0>(pipe);
//        __syncthreads();
//
//        // These fragments will accumulate the result of A and B matrix fragment multiplications
//        // along the K_GLOBAL dimension.
//        wmma::fragment<wmma::accumulator, M, N, K, double> c[WARP_COL_TILES][WARP_ROW_TILES];
//
//        // Load the C matrix tiles into fragments from shared memory.
//#pragma unroll
//        for (int i = 0; i < WARP_COL_TILES; i++) {
//#pragma unroll
//            for (int j = 0; j < WARP_ROW_TILES; j++) {
//                const double *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * N + j * N;
//
//                wmma::load_matrix_sync(c[i][j], tile_ptr, SHMEM_STRIDE, C_LAYOUT);
//                // Scale the C matrix.
//#pragma unroll
//                for (int t = 0; t < c[i][j].num_elements; t++) {
//                    c[i][j].x[t] *= beta;
//                }
//            }
//        }
//
//        pipe.consumer_release();
//        // sync here so that shared memory can then be used for loading A & B matrices.
//        __syncthreads();
//
//        // Select what warp copies what matrix to shared memory.
//        // Warps 0-3 copy the A matrix, warps 4-7 copy the B matrix.
//        const double *warp_ptr = (warpId < (WARPS_PER_BLOCK/2)) ? (&A[block_tile_i * M * K_GLOBAL] + M * K_GLOBAL * (warpId % (WARPS_PER_BLOCK/2)) * 2) :
//                                              (&B[block_tile_j * N * K_GLOBAL] + N * K_GLOBAL * (warpId % (WARPS_PER_BLOCK/2)) * 2);
//
//        const int stridePerLaneCopy = (laneId / CHUNK_COPY_LINE_LANES);
//        constexpr int chunksPerLane = ((WARP_SIZE/2) / CHUNK_COPY_LINES_PER_WARP);
//        const int laneLoadElem = (laneId % CHUNK_COPY_LINE_LANES) << loadStride;
//
//        // Go through the global K dimension by a fixed step at a time.
//#pragma unroll
//        for (int tile_k = 0; tile_k < K_TILES; tile_k += CHUNK_K) {
//            // Copy slices of the A and B matrices to shared memory.
//            // The first half of the warps in the CTA copy the A matrix, the rest copy the B matrix.
//            // As for DMMA  M == N we use M for warp 4-7 + shmem_idx_b_off.
//            size_t shmem_idx = (M * (warpId % (WARPS_PER_BLOCK/2)) * 2) + (shmem_idx_b_off * (warpId/(WARPS_PER_BLOCK/2)));
//
//            // First half of the warp copies the first row / column of the matrix,
//            // the second half of the warp copies the next.
//            const double *lane_ptr = warp_ptr + tile_k * K + stridePerLaneCopy * K_GLOBAL + laneLoadElem;
//
//            // Shift the second half of the warp to the next row / column in the shared memory.
//            shmem_idx += stridePerLaneCopy;
//#pragma unroll
//            for(int i = 0; i < chunksPerLane; i++) {
//                 // Copy 16 bytes at once in each lane.
//                pipe.producer_acquire();
//
//                cuda::memcpy_async(&shmem[shmem_idx][laneLoadElem], lane_ptr, shape2, pipe);
//
//                pipe.producer_commit();
//
//                // Advance the global memory pointer and the shared memory index.
//                lane_ptr = lane_ptr + K_GLOBAL * CHUNK_COPY_LINES_PER_WARP;
//                shmem_idx += CHUNK_COPY_LINES_PER_WARP;
//            }
//
//            cuda::pipeline_consumer_wait_prior<0>(pipe);
//            __syncthreads();
//
//            // Compute a grid of C matrix tiles in each warp.
//#pragma unroll
//            for (int k_step = 0; k_step < CHUNK_K; k_step++) {
//                wmma::fragment<wmma::matrix_a, M, N, K, double, wmma::row_major> a[WARP_COL_TILES];
//                wmma::fragment<wmma::matrix_b, M, N, K, double, wmma::col_major> b[WARP_ROW_TILES];
//#pragma unroll
//                for (int i = 0; i < WARP_COL_TILES; i++) {
//                    size_t shmem_idx_a = (warpId/2) * M * 2 + (i * M);
//                    const double *tile_ptr = &shmem[shmem_idx_a][k_step * K];
//
//                    wmma::load_matrix_sync(a[i], tile_ptr, K * CHUNK_K + SKEW_DOUBLE);
//#pragma unroll
//                    for (int j = 0; j < WARP_ROW_TILES; j++) {
//                        if (i == 0) {
//                            // Load the B matrix fragment once, because it is going to be reused
//                            // against the other A matrix fragments.
//                            size_t shmem_idx_b = shmem_idx_b_off + (WARP_ROW_TILES * N) * (warpId%2) + (j * N);
//                            const double *tile_ptr = &shmem[shmem_idx_b][k_step * K];
//
//                            wmma::load_matrix_sync(b[j], tile_ptr, K * CHUNK_K + SKEW_DOUBLE);
//                        }
//                        wmma::mma_sync(c[i][j], a[i], b[j], c[i][j]);
//                    }
//                }
//            }
//            pipe.consumer_release();
//            __syncthreads();
//        }
//
//        // Store the D fragments to shared memory.
//#pragma unroll
//        for (int i = 0; i < WARP_COL_TILES; i++) {
//#pragma unroll
//            for (int j = 0; j < WARP_ROW_TILES; j++) {
//                // Uniform, point-wise transformations of ALL fragment elements by ALL threads in the
//                // warp are well-defined even though element indices within fragment storage are not defined.
//#pragma unroll
//                for (int t = 0; t < c[i][j].num_elements; t++)
//                    c[i][j].x[t] *= alpha;
//
//                double *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * N + j * N;
//
//                wmma::store_matrix_sync(tile_ptr, c[i][j], SHMEM_STRIDE, C_LAYOUT);
//            }
//        }
//
//        __syncthreads();
//
//        // Now that shared memory contains all the D tiles, stream them to global memory.
//        double *dst_gmem_warp_stream_ptr = &D[gmem_idx];
//
//#pragma unroll
//        for (int i = 0; i < N; i++) {
//            *((int4*)(dst_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) + laneId) =
//                *((int4*)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId);
//        }
//
//        __syncthreads();
//    }
//#endif
//}
//
//__global__ void compute_dgemm_cg_async_copy(const double *A, const double *B, const double *C, double *D, double alpha, double beta)
//{
//#if __CUDA_ARCH__ >= 800
//    extern __shared__ double shmem[][CHUNK_K * K + SKEW_DOUBLE];
//    auto cta = cg::this_thread_block();
//    auto tile32 = cg::tiled_partition<32>(cta);
//
//    constexpr int tileChunkCopySize = WARP_SIZE / CHUNK_COPY_LINES_PER_WARP;
//    auto tileChunkCopy = cg::tiled_partition<tileChunkCopySize>(cta);
//
//    // Warp and lane identification.
//    const unsigned int warpId = threadIdx.x / WARP_SIZE;
//    const unsigned int laneId = threadIdx.x % WARP_SIZE;
//
//    // Offset in shared memory from which the B matrix is stored.
//    constexpr size_t shmem_idx_b_off = BLOCK_COL_TILES * M;
//
//    // This pointer is used to access the C and D matrix tiles this warp computes.
//    double *shmem_warp_tile_ptr = (double*)&shmem[0][0] + (warpId/2) * SHMEM_STRIDE * N * 2 + (warpId%2) * SHMEM_OFFSET;
//
//    // This pointer is used to stream the C and D matrices block-wide tile to and from shared memory.
//    double *shmem_warp_stream_ptr = (double*)&shmem[0][0] + warpId * SHMEM_STRIDE * N;
//
//    // Adjust the beta scaler, as it'll be multiplied by alpha at the end of
//    // each tile computation. Technically this is not generally correct (may result
//    // in a loss of precision). Zero still needs to be specially handled though.
//    beta /= alpha;
//
//    // Each CTA slides along the 64 x 64 tiles from the top left corner of the matrix to the
//    // right and down, and selects the next tile to compute. Once there's no such tile,
//    // all warps in this CTA exit.
//    for(unsigned int block_pos = blockIdx.x;; block_pos += gridDim.x) {
//        const unsigned int block_tile_i = ((block_pos * BLOCK_ROW_TILES) / N_TILES) * (BLOCK_COL_TILES);
//        const unsigned int block_tile_j = (block_pos * BLOCK_COL_TILES) % N_TILES;
//
//        // Stop when there are no more D matrix tiles to compute in this CTA.
//        if (block_tile_i >= M_TILES) {
//            break;
//        }
//
//        // This warp's pointer to the C matrix data to copy memory from to shared memory.
//        const size_t gmem_idx = (block_tile_i + warpId) * M * GLOBAL_MEM_STRIDE + block_tile_j * N;
//        const double *src_gmem_warp_stream_ptr = &C[gmem_idx];
//
//        // Stream multiple C tiles to shared memory.
//#pragma unroll
//        for (int i = 0; i < N; i++) {
//            auto dst_ptr = &shmem_warp_stream_ptr[(SHMEM_STRIDE * i)];
//            auto src_ptr = &src_gmem_warp_stream_ptr[(GLOBAL_MEM_STRIDE * i)];
//            cg::memcpy_async(tile32, dst_ptr, src_ptr, cuda::aligned_size_t<alignof(double2)>{tile32.size() * sizeof(double2)});
//        }
//
//        cg::wait(cta);
//
//        // These fragments will accumulate the result of A and B matrix fragment multiplications
//        // along the K_GLOBAL dimension.
//        wmma::fragment<wmma::accumulator, M, N, K, double> c[WARP_COL_TILES][WARP_ROW_TILES];
//
//        // Load the C matrix tiles into fragments from shared memory.
//#pragma unroll
//        for (int i = 0; i < WARP_COL_TILES; i++) {
//#pragma unroll
//            for (int j = 0; j < WARP_ROW_TILES; j++) {
//                const double *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * N + j * N;
//                wmma::load_matrix_sync(c[i][j], tile_ptr, SHMEM_STRIDE, C_LAYOUT);
//            }
//        }
//
//        // Scale the C matrix.
//#pragma unroll
//        for (int i = 0; i < WARP_COL_TILES; i++) {
//#pragma unroll
//            for (int j = 0; j < WARP_ROW_TILES; j++) {
//#pragma unroll
//                for (int t = 0; t < c[i][j].num_elements; t++) {
//                    c[i][j].x[t] *= beta;
//                }
//            }
//        }
//
//        // sync here so that shared memory can then be used for loading A & B matrices.
//        cg::wait(cta);
//        // Select what warp copies what matrix to shared memory.
//        // Warps 0-3 copy the A matrix, warps 4-7 copy the B matrix.
//        const double *warp_ptr = (warpId < 4) ? (&A[block_tile_i * M * K_GLOBAL] + M * K_GLOBAL * (warpId % (WARPS_PER_BLOCK/2)) * 2) :
//            (&B[block_tile_j * N * K_GLOBAL] + N * K_GLOBAL * (warpId % (WARPS_PER_BLOCK/2)) * 2);
//
//        const int stridePerLaneCopy = (laneId / CHUNK_COPY_LINE_LANES);
//        constexpr int chunksPerLane = ((WARP_SIZE/2) / CHUNK_COPY_LINES_PER_WARP);
//        // Go through the global K dimension by a fixed step at a time.
//#pragma unroll
//        for (int tile_k = 0; tile_k < K_TILES; tile_k += CHUNK_K) {
//            // Copy slices of the A and B matrices to shared memory.
//            // The first half of the warps in the CTA copy the A matrix, the rest copy the B matrix.
//            // As for DMMA  M == N we use M for warp 4-7 + shmem_idx_b_off.
//            size_t shmem_idx = (M * (warpId % (WARPS_PER_BLOCK/2)) * 2) + (shmem_idx_b_off * (warpId/(WARPS_PER_BLOCK/2)));
//
//            // First half of the warp copies the first row / column of the matrix,
//            // the second half of the warp copies the next.
//            auto lane_ptr = warp_ptr + tile_k * K + stridePerLaneCopy * K_GLOBAL;
//
//            // Shift the second half of the warp to the next row / column in the shared memory.
//            shmem_idx += stridePerLaneCopy;
//
//#pragma unroll
//            for(int i = 0; i < chunksPerLane; i++) {
//                // Copy 16 bytes at once in each lane.
//                auto dst_ptr = &shmem[shmem_idx][0];
//                auto src_ptr = lane_ptr;
//
//                cg::memcpy_async(tileChunkCopy, dst_ptr, src_ptr,
//                                cuda::aligned_size_t<alignof(double2)>{tileChunkCopySize * sizeof(double2)});
//
//                // Advance the global memory pointer and the shared memory index.
//                lane_ptr = lane_ptr + K_GLOBAL * CHUNK_COPY_LINES_PER_WARP;
//                shmem_idx += CHUNK_COPY_LINES_PER_WARP;
//            }
//            cg::wait(cta);
//
//            // Compute a grid of C matrix tiles in each warp.
//#pragma unroll
//            for (int k_step = 0; k_step < CHUNK_K; k_step++) {
//                wmma::fragment<wmma::matrix_a, M, N, K, double, wmma::row_major> a[WARP_COL_TILES];
//                wmma::fragment<wmma::matrix_b, M, N, K, double, wmma::col_major> b[WARP_ROW_TILES];
//
//#pragma unroll
//                for (int i = 0; i < WARP_COL_TILES; i++) {
//                    size_t shmem_idx_a = (warpId/2) * M * 2 + (i * M);
//                    const double *tile_ptr = &shmem[shmem_idx_a][k_step * K];
//
//                    wmma::load_matrix_sync(a[i], tile_ptr, K * CHUNK_K + SKEW_DOUBLE);
//
//#pragma unroll
//                    for (int j = 0; j < WARP_ROW_TILES; j++) {
//                        if (i == 0) {
//                            // Load the B matrix fragment once, because it is going to be reused
//                            // against the other A matrix fragments.
//                            size_t shmem_idx_b = shmem_idx_b_off + (WARP_ROW_TILES * N) * (warpId%2) + (j * N);
//                            const double *tile_ptr = &shmem[shmem_idx_b][k_step * K];
//
//                            wmma::load_matrix_sync(b[j], tile_ptr, K * CHUNK_K + SKEW_DOUBLE);
//
//                        }
//
//                        wmma::mma_sync(c[i][j], a[i], b[j], c[i][j]);
//                    }
//                }
//            }
//            cg::sync(cta);
//        }
//
//        // Store the D fragments to shared memory.
//#pragma unroll
//        for (int i = 0; i < WARP_COL_TILES; i++) {
//#pragma unroll
//            for (int j = 0; j < WARP_ROW_TILES; j++) {
//                // Uniform, point-wise transformations of ALL fragment elements by ALL threads in the
//                // warp are well-defined even though element indices within fragment storage are not defined.
//#pragma unroll
//                for (int t = 0; t < c[i][j].num_elements; t++)
//                    c[i][j].x[t] *= alpha;
//
//                double *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * N + j * N;
//
//                wmma::store_matrix_sync(tile_ptr, c[i][j], SHMEM_STRIDE, C_LAYOUT);
//            }
//        }
//
//        cg::sync(cta);
//
//        // Now that shared memory contains all the D tiles, stream them to global memory.
//        double *dst_gmem_warp_stream_ptr = &D[gmem_idx];
//
//#pragma unroll
//        for (int i = 0; i < N; i++) {
//            *((int4*)(dst_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) + laneId) =
//                *((int4*)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId);
//        }
//        cg::sync(cta);
//    }
//#endif
//}

// Performs an MxNxK DGEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 8, 8 and 4 respectively.
//  3) A is row major, B is column major matrix.
// Note: This is a less performant version of the compute_dgemm kernel. It is designed for
//       demonstration purposes only to show the CUDA WMMA API use without relying on
//       availability of the shared memory.
__global__ void simple_wmma_gemm(double *a, double *b, double *c, double *d, int m_ld, int n_ld, int k_ld, double alpha, double beta)
{
#if __CUDA_ARCH__ >= 800
    // Leading dimensions. Packed with no transpositions.
    int lda = k_ld;
    int ldb = k_ld;
    int ldc = n_ld;

    // Tile using a 2D grid
    int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
    int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

    // Declare the fragments
    wmma::fragment<wmma::matrix_a, M, N, K, double, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, M, N, K, double, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, M, N, K, double> acc_frag;
    wmma::fragment<wmma::accumulator, M, N, K, double> c_frag;

    wmma::fill_fragment(acc_frag, 0.0f);

    // Loop over k
    for (int i = 0; i < k_ld; i += K) {
        int aCol = i;
        int aRow = warpM * M;

        int bCol = warpN * N;
        int bRow = i;

        // Bounds checking
        if (aRow < m_ld && aCol < k_ld && bRow < k_ld && bCol < n_ld) {
            // Load the inputs
            wmma::load_matrix_sync(a_frag, a + aCol + aRow * lda, lda);
            wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);

            // Perform the matrix multiplication
            wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
        }
    }

    // Load in the current value of c, scale it by beta, and add this our result scaled by alpha
    int cCol = warpN * N;
    int cRow = warpM * M;

    if (cRow < m_ld && cCol < n_ld) {
        wmma::load_matrix_sync(c_frag, c + cCol + cRow * ldc, ldc, wmma::mem_row_major);

        for(int i=0; i < c_frag.num_elements; i++) {
            c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
        }

        // Store the output
        wmma::store_matrix_sync(d + cCol + cRow * ldc, c_frag, ldc, wmma::mem_row_major);
    }
#endif
}

__host__ void matMultiplyOnHost(double *A, double *B, double *C,
                                float alpha, float beta,
                                int numARows, int numAColumns,
                                int numBRows, int numBColumns,
                                int numCRows, int numCColumns)
{
    for (int i = 0; i < numCRows; i++) {
        for (int j = 0; j < numCColumns; j++) {
            double temp = 0.0;

            for (int k = 0; k < numAColumns; k++) {
                // B matrix is column major. A matrix is row major.
                temp += A[i * numAColumns + k] * B[j * numBRows + k];
            }

//            C[i*numCColumns + j] = temp * alpha + beta * C[i * numCColumns + j];
            C[i*numCColumns + j] = temp + C[i * numCColumns + j];
        }
    }
}

#endif // COMPUTE_PREC == 64